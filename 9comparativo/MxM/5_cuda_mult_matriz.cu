#include <stdio.h>
#include <hip/hip_runtime.h>
//#include "tempo.h"

const int L = 3; //8000; 
const int C = 3; //8000;


const int TAM_MATRIZ = L * C;

const int BLOCK_SIZE = 3;


void mostra_matriz(int mat[]);
void inicializa_matrizL(int mat[]);
void inicializa_matrizC(int mat[]);

__global__ void matrixMul(int* A, int* B, int* C)
{
	//Using blocks
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	for(int k = 0; k < L; ++k){
		C[i * L + j] += A[i * L + k] * B[k * L + j];
	}
}

int main(int argc, char** argv)
{

	int* h_A;
	int* h_B;
	int* h_C;

	int* d_A;
	int* d_B;
	int* d_C;
	
	size_t size = TAM_MATRIZ * sizeof(int);

	//alocação memoria RAM
	h_A = (int*)malloc(size);
	h_B = (int*)malloc(size);
	h_C = (int*)malloc(size);

	//inicializa matrizes
	inicializa_matrizL(h_A);
	inicializa_matrizC(h_B);

        printf("\n  M A T R I Z    A \n");
	mostra_matriz(h_A);

        printf("\n  M A T R I Z    B \n");
	mostra_matriz(h_B);

        printf("\n  M A T R I Z    C \n");
	mostra_matriz(h_C);

	//alocação memoria GPU
	hipMalloc((int**)&d_A, size);
	hipMalloc((int**)&d_B, size);
	hipMalloc((int**)&d_C, size);

	//copia matrizes da RAM para GPU
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

	//tempo1();

	//Call kernels
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(L / dimBlock.x, C / dimBlock.y);
	matrixMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	//tempo2();

	//copia matriz C da GPU para RAM
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


        printf("\n\n\n  M A T R I Z    C \n");
	mostra_matriz(h_C);

	//tempoFinal("mili segundos", argv[0], MSGLOG);

	//libera memorias
	free(h_A);
	free(h_B);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}

void inicializa_matrizL(int mat[])
{
        int z, v;
        printf("\n inicializando matriz ...\n");
        for (z=0; z < L; z ++)
		for (v=0; v < C; v++){
			mat[z * C + v] = z+1;
		}
}

void inicializa_matrizC(int mat[])
{
        int z, v;
        printf("\n inicializando matriz ...\n");
        for (z=0; z < L; z ++)
		for (v=0; v < C; v++){
			mat[z * C + v] = v+2;
		}
}

void mostra_matriz(int mat[])
{
        int z, v;
        for (z=0; z < L; z ++){
		for (v=0; v < L; v++) {
        	        printf(" %4d", mat[z * L + v]);
		}
        	printf("\n");
	}
}
