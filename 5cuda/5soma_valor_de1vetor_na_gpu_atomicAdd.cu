#include <stdio.h>
#include <hip/hip_runtime.h>

#define TAM 10


//-----aloca memória global - na RAM e na GPU
__managed__ int vetor[TAM];




//------Kernel que sera executado na GPU
__global__ void incrementa_vetor(int *s)
{
  	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	printf("\n GPU --- somando posição [%d] ", idx);
	atomicAdd(s, vetor[idx]);
}




int main(int argc, char *argv[0])
{

	int i;

	//-----cria uma var do tipo ponteiro
	int *soma;
	//-----aloca 4 bytes na memória RAM e na GPU
	//4 é qtde de bytes de um inteiro = sizeof(int)
  	hipMallocManaged(&soma, 4); 
	//-----inicializa endereço do ponteiro com 0
	*soma = 0;


	// Initializa vetor na CPU
	for (i=0; i<TAM; i++){ 
		vetor[i] =  i;			
	}

	//-----programa principal exibe o vetor inicial
	for (i=0; i<TAM; i++){ 
		printf("\n CPU --- vet[%2d] = %2d", i, vetor[i]);
	}
	printf("\n");

 							
	//-----executa a função incrementa_vetor na GPU com 10 threads 
	incrementa_vetor <<< 2, 5 >>> (soma);
	
	//-----cria uma barreira - espera todas as threads finalizarem
	hipDeviceSynchronize();
 

	//-----programa principal exibe o vetor resultado	
	printf("\n\n CPU --- soma = %d ", *soma);

	printf("\n");

}




