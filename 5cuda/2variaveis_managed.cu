
#include <hip/hip_runtime.h>
#include <stdio.h>

#define L 10


__managed__ int vetor[L];

        // managed - Memória unificada 
        //- os dados sao alocados nas e mem (RAM e GPU)
        //- os dados sao sincronizados entre as mem





__global__ void inicializa_vetor()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
        printf("\n GPU - inicializando - L= %d\n", idx);
        
        vetor[idx] = idx;

}



__global__ void dobro()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        printf("\n GPU - calculando - th %d \n", idx);
        
        vetor[idx] = vetor[idx] * 2;

}


__host__ void mostra_vetor()
{
        int z;
        printf("\n CPU - V E T O R \n");
        for (z=0; z < L; z ++){
                printf("  %3d", vetor[z]);
        }
  printf("\n\n\n");
}









int main(int argc, char *argv[])
{
    

    inicializa_vetor <<< 2, 5 >>> ();
    hipDeviceSynchronize();  
    mostra_vetor();


    dobro <<< 2, 5 >>> ();
    hipDeviceSynchronize();
    mostra_vetor();



}












/*
 

 ## Qualificadores de tipo de variável

__device__ 
  - define uma variável no espaço de memória global da GPU 
  - ela é acessível por todas as threads de um grid e também a partir da CPU  
  - tempo de vida é o da aplicação  

__constant__ 
  - define uma variável no espaço de memória constante da GPU

__shared__ 
  - define uma uma var na memória compartilhada da GPU
  - ela é acessível apenas para as threads do mesmo bloco
  - tempo de vida é o do bloco

__managed__ 
  - permite o uso de "Memória unificada"



In terms of speed:

1st : Register file
2nd : Shared Memory
3rd : Constant Memory
4th : Texture Memory
Last place: Local Memory and Global Memory

  
   */