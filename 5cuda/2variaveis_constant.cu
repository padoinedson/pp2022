#include "hip/hip_runtime.h"
#include <stdio.h>
#include "tempo.h"





 

__device__ __constant__  int vetor[4] = {1,2,3,4};


__global__ void mostra()
{

  printf("%d\n", vetor[3]);
}




int main(int argc, char *argv[])
{
    
  mostra<<<1,1>>>();

  hipDeviceSynchronize();

}












/*
 

 ## Qualificadores de tipo de variável

__device__ 
  - define uma variável no espaço de memória global da GPU 
  - ela é acessível por todas as threads de um grid e também a partir da CPU  
  - tempo de vida é o da aplicação  

__constant__ 
  - define uma variável no espaço de memória constante da GPU
  - constant memory only accessible on the device

__shared__ 
  - define uma uma var na memória compartilhada da GPU
  - ela é acessível apenas para as threads do mesmo bloco
  - tempo de vida é o do bloco

__managed__ 
  - permite o uso de "Memória unificada"


  
   */




/*
#include <stdio.h>

__device__ float c_array[10];
 
__global__ void kernel(float* d_array)
{ 

    d_array[0] = c_array[0]; 
}
 
int main()
{

    float* d_array = 0;
    
    float h_array[10] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};   
    
    hipMalloc((void**)&d_array, 10 * sizeof(float));
    
    hipMemcpyToSymbol(HIP_SYMBOL("c_array"), h_array, sizeof(float)*10, 0, hipMemcpyHostToDevice );
    
    kernel<<< 1, 1 >>>(d_array);

}


=========================



 __constant__ int soma;
 

__global__ void mostra2()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
        printf("\n GPU - varconstant = %d\n", idx);
        
        int N = 2<<20;
        
        int x =0;

        
        for (int i= 0; i < N; i++) 
            x += soma;


        printf("\n GPU - soma = %d\n", soma);

}




  int h_soma = 10;   
  hipMemcpyToSymbol(HIP_SYMBOL(soma), h_soma, sizeof(int)*1, 0, hipMemcpyHostToDevice );
  mostra<<<1,1>>>();
  hipDeviceSynchronize();

*/