

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printNumber()
{
  printf(" GPU - iteration= %d\n", threadIdx.x);
  
}


int main()
{

  printNumber<<<1, 5>>>();

  hipDeviceSynchronize();

}


