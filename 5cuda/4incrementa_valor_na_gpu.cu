#include <stdio.h>
#include <hip/hip_runtime.h>




//------Kernel que sera executado na GPU
__global__ void incrementa_valor(int *num)
{
  	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	
	printf("\n GPU - thread numero = %d  --  num = %d", idx, num);

	atomicAdd(num, 1);
}




int main(int argc, char *argv[0])
{
	//-----cria uma var do tipo ponteiro
	int *num;


	//-----aloca 4 bytes na memória RAM e na GPU
	//4 é qtde de bytes de um inteiro = sizeof(int)
  	hipMallocManaged(&num, 4);   


	//-----inicializa endereço do ponteiro com 0
  	*num = 0;

	//-----exibe valor inicial
	printf("\n CPU - programa iniciou - num = %d \n\n", *num);

	//-----executa a função incrementa_valor na GPU com 4 threads 
	incrementa_valor <<< 2, 2 >>> (num);

	//-----cria uma barreira - espera todas as threads finalizarem
	hipDeviceSynchronize();
 
	//-----exibe resultado
	printf("\n\n\n CPU - programa finalizou - num = %d \n\n\n", *num);
 }


