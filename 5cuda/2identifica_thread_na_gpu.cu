
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printNumber()
{

            //threadIdx.x numero da th dentro do bloco
    printf("\n GPU - thread numero = %d  foi criada", threadIdx.x);

}


int main()
{

    printf("\n CPU - programa iniciou");

    printNumber<<<2, 4>>>();

    hipDeviceSynchronize();

    printf("\n CPU - programa finalizou\n\n");
}



//video 3 and 4


