// quantidade de thread < que o tam dos dados


#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  /*
   * Use a grid-stride loop so each thread does work
   * on more than one element in the array.
   */

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] *= 2;
    printf ("\n GPU - kernel %d - stride %d index %d value %d", idx, stride, i, a[i]);
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 32;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  size_t threads_per_block = 4;
  size_t number_of_blocks = 2;


  printf("\n CPU - N = %d \n", N);

  printf("\n CPU - numberOfBlocks = %ld\n", number_of_blocks);

  printf("\n CPU - threadPerBlock = %ld\n", threads_per_block);

  printf("\n CPU - totalThread = %ld\n\n", threads_per_block*number_of_blocks);





  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("\n CPU - All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}


//video 6