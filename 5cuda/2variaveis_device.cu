#include "hip/hip_runtime.h"
#include <stdio.h>
#include "tempo.h"


#define L 10
 
__device__ int vetor[L];

 
 



__global__ void inicializa_vetor()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
        printf("\n GPU - inicializando - L= %d\n", idx);
        
        vetor[idx] = idx;

}



__global__ void dobro()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        printf("\n GPU - calculando - th %d \n", idx);
        
        vetor[idx] = vetor[idx] * 2;

}




__global__ void mostra_vetor()
{
        int z;
        printf("\n GPU - V E T O R \n");
        for (z=0; z < L; z ++){
                printf("  %3d", vetor[z]);
        }
  printf("\n\n\n");
}

 
 





int main(int argc, char *argv[])
{
    
    int *deviceArray = 0;
    
    //aloca mem na GPU
    hipMalloc(&deviceArray, L * sizeof(int));



    inicializa_vetor <<< 2, 5 >>> ();
    hipDeviceSynchronize();  
    mostra_vetor <<< 1, 1 >>>();


    dobro <<< 2, 5 >>> ();
    hipDeviceSynchronize();
    mostra_vetor <<< 1, 1 >>>();

}












/*
 

 ## Qualificadores de tipo de variável

__device__ 
  - define uma variável no espaço de memória global da GPU 
  - ela é acessível por todas as threads de um grid e também a partir da CPU  
  - tempo de vida é o da aplicação  

__constant__ 
  - define uma variável no espaço de memória constante da GPU

__shared__ 
  - define uma uma var na memória compartilhada da GPU
  - ela é acessível apenas para as threads do mesmo bloco
  - tempo de vida é o do bloco

__managed__ 
  - permite o uso de "Memória unificada"




In terms of speed:

1st : Register file
2nd : Shared Memory
3rd : Constant Memory
4th : Texture Memory
Last place: Local Memory and Global Memory

  
   */