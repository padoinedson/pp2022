
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>



//------Kernel que sera executado na GPU
__global__ void mostra1()
{
 
	printf("\n thread numero = %d  foi criada", threadIdx.x);

}




int main(int argc, char *argv[0])
{


 	printf("\n CPU - programa iniciou \n\n\n");

		//-----executa a função na GPU com n threads 
		//----- cria n blocos com n threads em cada bloco
	mostra1 <<< 2, 5 >>> ();


		//-----cria uma barreira - espera todas as threads finalizarem
	hipDeviceSynchronize();
 

  	printf("\n\n\n CPU - programa finalizou \n\n\n");



 }


