
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

void CPUFunction()
{
  printf("CPU - rodando ...\n");
}

__global__ void GPUFunction()
{
  printf("GPU - rodando ....\n");
}

int main()
{
  CPUFunction();

  GPUFunction<<<1, 1>>>();

  hipDeviceSynchronize();
}














  /*
   * The addition of `__global__` signifies that this function
   * should be launced on the GPU.
   */


  /*
   * Add an execution configuration with the <<<...>>> syntax
   * will launch this function as a kernel on the GPU.
   */


  /*
   * `cudaDeviceSynchronize` will block the CPU stream until
   * all GPU kernels have completed.
   */ 