
#include <stdio.h>
#include <hip/hip_runtime.h>


//------Kernel que sera executado na GPU
__global__ void incrementa_valor()
{
           //calcula numero da th geral
           //num_bloco * qtde th_do_bloco + id_th_dentro_bloco

  int idx = blockIdx.x * blockDim.x       + threadIdx.x;
    
  printf("\n GPU - thread id-geral= %d  - id-bloco= %d  - id-dentro-bloco= %d\n ", idx, blockIdx.x, threadIdx.x);

}




int main(int argc, char *argv[0])
{

  printf("\n CPU - programa iniciou \n\n\n");

  //-----executa a função incrementa_valor na GPU com 4 threads 
  //----- cria 2 blocos com 2 threads em cada bloco
  incrementa_valor <<< 2, 2 >>> ();

  //-----cria uma barreira - espera todas as threads finalizarem
  hipDeviceSynchronize();
 
  //-----exibe resultado
  printf("\n\n\n CPU - programa finalizou \n\n\n");

 }


