
#include <hip/hip_runtime.h>
#include <stdio.h>


int main()
{

  int deviceId;
  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);


  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;

  printf("  Device ID: %d\n   Number of SMs: %d\n   Compute Capability Major: %d\n   Compute Capability Minor: %d\n   Warp Size: %d\n",   deviceId,  multiProcessorCount,    computeCapabilityMajor,    computeCapabilityMinor,    warpSize);


}


