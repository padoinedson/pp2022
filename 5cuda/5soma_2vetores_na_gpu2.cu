#include "hip/hip_runtime.h"
//versao 2 - qtde de dados < qtde de threads - 7 < 10
//limite


#include <stdio.h>
#include <hip/hip_runtime.h>

#define TAM 7


//-----aloca memória global - na RAM e na GPU
__managed__ int vetorA[TAM];
__managed__ int vetorB[TAM];
__managed__ int vetorC[TAM];


//------Kernel que sera executado na GPU
__global__ void soma_vetor()
{
  	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < TAM){   // limita a qtde de threads
		vetorC[idx] = vetorA[idx] + vetorB[idx];
		printf("\n GPU - thread = %d  bloco = %d somando ", idx, blockIdx.x);
	}
}




int main(int argc, char *argv[0])
{

    srand(time(NULL));
    
	int i;
 

	// Initializa vetor na CPU
	for (i=0; i<TAM; i++){ 
		vetorA[i] =  rand() % 100;		
		vetorB[i] =  rand() % 100;	
	}

	//-----programa principal exibe o vetor inicial
	for (i=0; i<TAM; i++){ 
		printf("\n CPU - i = %d - vetorA = %2d  -  vetorB = %2d", i, vetorA[i], vetorB[i]);
	}
	printf("\n");
				
	//-----executa a função incrementa_vetor na GPU com 10 threads 
	soma_vetor <<< 2, 5 >>> ();
	
	//-----cria uma barreira - espera todas as threads finalizarem
	hipDeviceSynchronize();
 

	//-----programa principal exibe o vetor resultado	
	for (i=0; i<TAM; i++){ 
		printf("\n CPU - i = %d - vetorC = %4d", i, vetorC[i]);
	}
	printf("\n");

}




