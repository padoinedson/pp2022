
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>  // rand(), srand()
//#include "tempo.h"


#define L 10

__managed__ int vetor[L];


__global__ void inicializa_vetor()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
        printf("\n GPU - inicializando - L= %d\n", idx);
        
        vetor[idx] = idx;
}



__global__ void dobro()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        printf("\n GPU - calculando - th %d \n", idx);
        
        vetor[idx] = vetor[idx] * 2;

}


void mostra_vetor();



int main(int argc, char *argv[])
{

    //tempo1();
    

    inicializa_vetor <<< 2, 5 >>> ();
    hipDeviceSynchronize();  
    mostra_vetor();


    dobro <<< 2, 5 >>> ();
    hipDeviceSynchronize();
    mostra_vetor();

    //tempo2();

    //tempoFinal("mili segundos", argv[0], MSGLOG);

}


void mostra_vetor()
{
        int z;
        printf("\n CPU - V E T O R \n");
        for (z=0; z < L; z ++){
                printf("  %3d", vetor[z]);
        }
	printf("\n\n\n");
}










//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#thread-hierarchy




