
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>



__host__ void mostra()
{
 
 	printf("\n CPU - programa iniciou \n\n\n");  	

}



__global__ void mostra1()
{
  	
	printf("\n GPU - thread numero = %d  foi criada", threadIdx.x);
//	mostra2 <<< 1, 1 >>> ();

}



__device__ void mostra2()
{
  	
	printf("\n GPU - thread numero = %d  foi criada", threadIdx.x);

}





int main(int argc, char *argv[0])
{


	mostra();

	mostra1 <<< 1, 1 >>> ();

 	hipDeviceSynchronize();
  
  	printf("\n\n\n CPU - programa finalizou \n\n\n");

 }






/* Qualificadores de tipo de função


__global__  
	- define uma função que é executada na GPU 
	- é chamada a partir da CPU  

__device__  
	- define uma função que será executada na GPU 
	- só pode ser chamada a partir da GPU  

__host__ 
	- define uma função que será executada na CPU 
	- só pode ser chamada a partir da CPU  


*/