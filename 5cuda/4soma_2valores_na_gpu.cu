#include <stdio.h>
#include <hip/hip_runtime.h>




//------Kernel que sera executado na GPU
__global__ void soma_numeros(int *num1, int *num2)
{
  	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  	
  	*num1 = *num1 + *num2;
  	
	printf("\n GPU - thread numero = %d  --  num = %d", idx, *num1);

}




int main(int argc, char *argv[0])
{
	//-----cria uma var do tipo ponteiro
	int *num1, *num2;

	//-----aloca 4 bytes na memória RAM e na GPU
  	hipMallocManaged(&num1, 4);   //sizeof(int)
  	hipMallocManaged(&num2, 4);   //sizeof(int)

	//-----inicializa endereço do ponteiro com 0
  	*num1 = 4;
  	*num2 = 5;  	

	//-----exibe valor inicial
	printf("\n CPU - programa iniciou - num = %d \n\n", *num1);

	//-----executa a função incrementa_valor na GPU com 4 threads 
	soma_numeros <<< 1, 1 >>> (num1, num2);

	//-----cria uma barreira - espera todas as threads finalizarem
	hipDeviceSynchronize();
 
	//-----exibe resultado
	printf("\n\n\n CPU - programa finalizou - num = %d \n\n\n", *num1);
 }


