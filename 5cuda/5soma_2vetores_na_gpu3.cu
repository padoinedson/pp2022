#include "hip/hip_runtime.h"
//versao 3 - qtde de dados > qtde de threads = 15 > 10
//stride


#include <stdio.h>
#include <hip/hip_runtime.h>

#define TAM 15


//-----aloca memória global - na RAM e na GPU
__managed__ int vetorA[TAM];
__managed__ int vetorB[TAM];
__managed__ int vetorC[TAM];


//------Kernel que sera executado na GPU
__global__ void soma_vetor()
{
  	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;				

	for (int i = idx; i < TAM; i += stride)
	{	
		vetorC[i] = vetorA[i] + vetorB[i];
		printf("\n GPU - thread = %2d  - stride %d - bloco = %d somando ", idx, stride, blockIdx.x);	
	}
}




int main(int argc, char *argv[0])
{

    srand(time(NULL));
    
	int i;
 

	// Initializa vetor na CPU
	for (i=0; i<TAM; i++){ 
		vetorA[i] =  rand() % 100;		
		vetorB[i] =  rand() % 100;	
	}

	//-----programa principal exibe o vetor inicial
	for (i=0; i<TAM; i++){ 
		printf("\n CPU - i = %2d - vetorA = %2d  -  vetorB = %2d", i, vetorA[i], vetorB[i]);
	}
	printf("\n");
				
	//-----executa a função incrementa_vetor na GPU com 10 threads 
	soma_vetor <<< 2, 5 >>> ();
	
	//-----cria uma barreira - espera todas as threads finalizarem
	hipDeviceSynchronize();
 

	//-----programa principal exibe o vetor resultado	
	for (i=0; i<TAM; i++){ 
		printf("\n CPU - i = %2d - vetorC = %4d", i, vetorC[i]);
	}
	printf("\n");

}




