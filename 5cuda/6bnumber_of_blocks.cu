// usar o N como limitante no kernel da GPU
// calculando o numero de blocos 




#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N)
  {
    a[i] = initialValue;
  }
}

int main()
{
 

  int N = 1000;

  int *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);

  /*
   * Assume we have reason to want the number of threads
   * fixed at `256`: do not modify `threads_per_block`.
   */

  size_t threads_per_block = 256;

  /*
   * The following is idiomatic CUDA to make sure there are at
   * least as many threads in the grid as there are `N` elements.
   */




  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;



  printf("\n CPU - N = %d \n", N);

  printf("\n CPU - numberOfBlocks = %ld\n", number_of_blocks);

  printf("\n CPU - threadPerBlock = %ld\n", threads_per_block);

  printf("\n CPU - totalThread = %ld\n\n", threads_per_block*number_of_blocks);






  int initialValue = 6;

  initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
  hipDeviceSynchronize();


  for (int i = 0; i < N; ++i)
  {
    if(a[i] != initialValue)
    {
      printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
      hipFree(a);
      exit(1);
    }
  }
  printf("SUCCESS!\n");

  hipFree(a);
}

