#include "hip/hip_runtime.h"
//exemplo para comparar o tempo
//entre global e shared mem



#include <stdio.h>
#include "tempo.h"






__global__ void varshared()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
        printf("\n GPU - varshared = %d\n", idx);
        
        int N = 2<<20;
         __shared__ int soma;
        

        for (int i= 0; i < N; i++) 
            soma = i;


//        __syncthreads();

        printf("\n GPU - soma = %d\n", soma);


}



__global__ void varglobal()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        printf("\n GPU - varglobal - th %d \n", idx);
        
        int N = 2<<20;
        int soma;

        for (int i= 0; i < N; i++) 
            soma = i;

        printf("\n GPU - soma = %d\n", soma);

}

 







int main(int argc, char *argv[])
{
    
    tempo1();
    varglobal <<< 1, 1 >>> ();
    hipDeviceSynchronize();  
    tempo2();
    tempoFinal("mili segundos", argv[0], MSGLOG);



    tempo1();
    varshared <<< 1, 1 >>> ();
    hipDeviceSynchronize();
    tempo2();
    tempoFinal("mili segundos", argv[0], MSGLOG);

}












/*
 

 ## Qualificadores de tipo de variável

__device__ 
  - define uma variável no espaço de memória global da GPU 
  - ela é acessível por todas as threads de um grid e também a partir da CPU  
  - tempo de vida é o da aplicação  

__constant__ 
  - define uma variável no espaço de memória constante da GPU

__shared__ 
  - define uma uma var na memória compartilhada da GPU
  - ela é acessível apenas para as threads do mesmo bloco
  - tempo de vida é o do bloco

__managed__ 
  - permite o uso de "Memória unificada"


  
   */