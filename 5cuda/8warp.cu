#include "hip/hip_runtime.h"
// quantidade de thread < que o tam dos dados

#include <stdio.h>
#include "tempo.h"







void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}




__global__ void doubleElements(int *a, int N)
{


  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] *= 2;
    //printf ("\n GPU - stride %d index %d value %d", stride, i, a[i]);
  }
}





bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}






int main(int argc, char *argv[])
{
    int N = 2<<20;
    int *a;

    size_t size = N * sizeof(int);
    hipMallocManaged(&a, size);

    init(a, N);


    size_t threads_per_block = 256;
    size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;




        printf("\n CPU - N = %d \n", N);

        printf("\n CPU - numberOfBlocks = %ld\n", number_of_blocks);

        printf("\n CPU - threadPerBlock = %ld\n", threads_per_block);

        printf("\n CPU - totalThread = %ld\n\n", threads_per_block*number_of_blocks);



    tempo1();
    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    hipDeviceSynchronize();
    tempo2();
    tempoFinal("mili segundos", argv[0], MSGLOG);

    bool areDoubled = checkElementsAreDoubled(a, N);
    printf("\n CPU - All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");




 



  
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    
    

    threads_per_block = 256;
    number_of_blocks = 32 * numberOfSMs;



        printf("\n CPU - N = %d \n", N);

        printf("\n CPU - numberOfBlocks = %ld\n", number_of_blocks);

        printf("\n CPU - threadPerBlock = %ld\n", threads_per_block);

        printf("\n CPU - totalThread = %ld\n\n", threads_per_block*number_of_blocks);

    tempo1();
    doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
    hipDeviceSynchronize();
    tempo2();
    tempoFinal("mili segundos", argv[0], MSGLOG);

    areDoubled = checkElementsAreDoubled(a, N);
    printf("\n CPU - All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");





  hipFree(a);
}

 