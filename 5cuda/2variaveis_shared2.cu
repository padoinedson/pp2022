#include "hip/hip_runtime.h"
//exemplo analisar o acesso 
//entre global e shared mem



#include <stdio.h>
#include "tempo.h"



__managed__ int somaglobal;


__global__ void varshared()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
        //printf("\n GPU - varshared - th %d\n", idx);
        
        __shared__ int soma;
        

        if (idx == 0) soma = 0;

        soma += threadIdx.x;


        __syncthreads();

        printf("\n GPU - idx %d somashared = %d\n", idx, soma);


}



__global__ void varglobal()
{
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        //printf("\n GPU - varglobal - th %d \n", idx);
        
        //int soma;

        if (idx == 0) somaglobal = 0;        

        somaglobal += threadIdx.x;

        __syncthreads();
        
        printf("\n GPU - idx %d somaglobal = %d\n", idx, somaglobal);

}

 







int main(int argc, char *argv[])
{
    
    tempo1();
    varglobal <<< 2, 3 >>> ();
    hipDeviceSynchronize();  
    tempo2();
    tempoFinal("mili segundos", argv[0], MSGLOG);



    tempo1();
    varshared <<< 2, 3 >>> ();
    hipDeviceSynchronize();
    tempo2();
    tempoFinal("mili segundos", argv[0], MSGLOG);

}












/*
 

 ## Qualificadores de tipo de variável

__device__ 
  - define uma variável no espaço de memória global da GPU 
  - ela é acessível por todas as threads de um grid e também a partir da CPU  
  - tempo de vida é o da aplicação  

__constant__ 
  - define uma variável no espaço de memória constante da GPU

__shared__ 
  - define uma uma var na memória compartilhada da GPU
  - ela é acessível apenas para as threads do mesmo bloco
  - tempo de vida é o do bloco

__managed__ 
  - permite o uso de "Memória unificada"


  
   */