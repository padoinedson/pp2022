
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>  // rand(), srand()
//#include "tempo.h"


#define L 4
#define C L
#define P L

__managed__ int cubo[C][L][P];

__global__ void inicializa_cubo()
{        
        
        int idx =  threadIdx.x;
        int idy =  threadIdx.y;
        int idz =  threadIdx.z;

        printf("\n GPU - inicializando - L= %d C= %d P= %d \n", idx, idy, idz);

        cubo[idx][idy][idz] = idx + idy + idz;
}



__global__ void dobro()
{
        int idx =  threadIdx.x;
        int idy =  threadIdx.y;
        int idz =  threadIdx.z;

        printf("\n GPU -  calculando...\n");

        cubo[idx][idy][idz] *= 2;

}



void mostra_cubo();


int main(int argc, char *argv[])
{


    //tempo1();
    


    //kernel com um bloco de L * C * P * 1 threads
    int numBlocks = 1;
    dim3 threadsPerBlock(L, C, P);
  

    inicializa_cubo <<< numBlocks, threadsPerBlock >>> ();
    hipDeviceSynchronize();  
    mostra_cubo();

    dobro <<< numBlocks, threadsPerBlock >>> ();
    hipDeviceSynchronize();
    mostra_cubo();

    //tempo2();

    //tempoFinal("mili segundos", argv[0], MSGLOG);
}




void mostra_cubo()
{
    int z, v, x;
        printf("\n CPU - C U B O \n");

        for (z=0; z < L; z ++){
            for (v=0; v < C; v++) {
                for (x=0; x < P; x++){
                    printf(" %4d", cubo[z][v][x]);
            }
            printf("\n");
        }
        printf("\n");
    }
    printf("\n\n\n");
    
}






//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#thread-hierarchy

