// quantidade de thread > que o tam dos dados
// usar o N como limitante no kernel da GPU




#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N)
  {
    a[i] *= 2;
  }
}


bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}


int main()
{
  int N = 10;
  int *a;

  size_t size = N * sizeof(int);


  hipMallocManaged(&a, size);

  init(a, N);

  size_t threads_per_block = 3;
  size_t number_of_blocks = 4;


  printf("\n CPU - N = %d \n", N);

  printf("\n CPU - numberOfBlocks = %ld\n", number_of_blocks);

  printf("\n CPU - threadPerBlock = %ld\n", threads_per_block);

  printf("\n CPU - totalThread = %ld\n\n", threads_per_block*number_of_blocks);



  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf(" CPU - All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");



  hipFree(a);
}




//video 5

//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#thread-hierarchy
//On current GPUs, a thread block may contain up to 1024 threads