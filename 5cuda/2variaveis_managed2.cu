#include <stdio.h>
#include <hip/hip_runtime.h>



__global__ void dobro(float *a, int N)
{
  	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx<N) {
	    printf("\n GPU - calculando - th %d \n", idx);
		a[idx] = a[idx] * 2;
	}
}




int main(int argc, char *argv[0])
{

	int i;
	float *a_h;								// pointeiro da RAM 
	float *a_d;  							// pointeiro da GPU
	const int N = 10;  						// tamanho do vetor
	size_t size = N * sizeof(float);		// qtde de bytes do vetor = 40


	//------aloca memoria na CPU
	a_h = (float *)malloc(size);        		


	//------aloca memoria na GPU
	hipMalloc((void **) &a_d, size);   		


	// Initializa vetor na CPU
	for (i=0; i<N; i++){ 
		a_h[i] = (float)i+2;			
	}

	// exibe vetor	
	for (i=0; i<N; i++){ 
		printf("\n CPU i = %d - vetor =%3f", i, a_h[i]);
	}


	//------copia variaval da RAM para a GPU  
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
									

	//------define qtde de blocos e qtde de threads por bloco
	int block_size = 10;          	
	int n_blocks = 1;   	        
	printf("\n\nqtde de blocos:   %d",n_blocks);						
	printf("\ntamanho do bloco: %d\n\n", block_size);



	//------executa o kernel na GPU
	dobro <<< n_blocks, block_size >>> (a_d, N);


	//------copia variaval da GPU para a RAM
	hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);


	//-----exibe resultado	
	for (i=0; i<N; i++){ 
		printf("\n CPU i = %d - vetor =%3f", i, a_h[i]);
	}

    //---libera memória RAM
	free(a_h); 

    //---libera memória da GPU
	hipFree(a_d); 

}




