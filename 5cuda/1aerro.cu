
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

void CPUFunction()
{
  printf("CPU - rodando ....\n");
}

__global__ void GPUFunction()
{
  printf("CPU - rodando ....\n");
}

int main()
{
  CPUFunction();

  GPUFunction<<<1, 1>>>();

//  cudaDeviceSynchronize();
}


/*
   * sem a diretiva `cudaDeviceSynchronize` o programa principal não espera o Kernel que está rodando na GPU
*/ 