
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>  // rand(), srand()
//#include "tempo.h"


#define L 5
#define C L

__managed__ int matriz[C][L];


__global__ void inicializa_matriz()
{        
        

        int idx =  threadIdx.x;
        int idy =  threadIdx.y;

        printf("\n GPU - inicializando - L= %d C= %d \n", idx, idy);

        matriz[idx][idy] = idx;
}



__global__ void dobro()
{
        int idx =  threadIdx.x;
        int idy =  threadIdx.y;

        printf("\n GPU -  calculando...\n");

        matriz[idx][idy] *= 2;

}



void mostra_matriz();


int main(int argc, char *argv[])
{

    //tempo1();
    


    //kernel com um bloco de L * C * 1 threads
    int numBlocks = 1;
    dim3 threadsPerBlock(L, C);
  
    inicializa_matriz <<< numBlocks, threadsPerBlock >>> ();
    hipDeviceSynchronize();  
    mostra_matriz();

    dobro <<< numBlocks, threadsPerBlock >>> ();
    hipDeviceSynchronize();
    mostra_matriz();


    //tempo2();

    //tempoFinal("mili segundos", argv[0], MSGLOG);

}



void mostra_matriz()
{
        int z, v;
        printf("\n CPU - M A T R I Z \n");

        for (z=0; z < L; z ++){
		for (v=0; v < C; v++) {
        	        printf("  %3d", matriz[z][v]);
		}
        	printf("\n");
	}
	printf("\n\n\n");
	
}









//https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#thread-hierarchy

