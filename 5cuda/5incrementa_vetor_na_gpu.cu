#include <stdio.h>
#include <hip/hip_runtime.h>

#define TAM 10


//-----aloca memória global - na RAM e na GPU
__managed__ int vetor[TAM];




//------Kernel que sera executado na GPU
__global__ void incrementa_vetor()
{
  	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	vetor[idx] = vetor[idx] + 1000;
}




int main(int argc, char *argv[0])
{

	int i;

	// Initializa vetor na CPU
	for (i=0; i<TAM; i++){ 
		vetor[i] =  i;			
	}

	//-----programa principal exibe o vetor inicial
	for (i=0; i<TAM; i++){ 
		printf("\n CPU i = %2d - vetor = %2d", i, vetor[i]);
	}
	printf("\n");

 							
	//-----executa a função incrementa_vetor na GPU com 10 threads 
	incrementa_vetor <<< 2, 5 >>> ();
	
	//-----cria uma barreira - espera todas as threads finalizarem
	hipDeviceSynchronize();
 

	//-----programa principal exibe o vetor resultado	
	for (i=0; i<TAM; i++){ 
		printf("\n CPU i = %d - vetor = %d", i, vetor[i]);
	}
	printf("\n");

}




